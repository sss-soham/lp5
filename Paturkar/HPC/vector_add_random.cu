
#include <iostream>
#include <fstream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int *A, const int *B, int *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(int);

    std::vector<int> h_A(N), h_B(N), h_C(N);

    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < N; ++i) {
        h_A[i] = rand() % 101;  // integers from 0 to 100
        h_B[i] = rand() % 101;
    }

    int *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    checkCudaError(hipMalloc(&d_A, size), "Allocating d_A");
    checkCudaError(hipMalloc(&d_B, size), "Allocating d_B");
    checkCudaError(hipMalloc(&d_C, size), "Allocating d_C");

    checkCudaError(hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice), "Copying h_A");
    checkCudaError(hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice), "Copying h_B");

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution");

    checkCudaError(hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost), "Copying result");

    // Save to file in column format
    std::ofstream outFile("vector_sum_output.txt");
    if (!outFile.is_open()) {
        std::cerr << "Error opening output file!" << std::endl;
        return 1;
    }

    outFile << std::setw(10) << "A[i]"
            << std::setw(10) << "B[i]"
            << std::setw(15) << "C[i] = A + B" << "\n";
    outFile << std::string(35, '-') << "\n";

    for (int i = 0; i < N; ++i) {
        outFile << std::setw(10) << h_A[i]
                << std::setw(10) << h_B[i]
                << std::setw(15) << h_C[i] << "\n";
    }

    outFile.close();

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}

